#include "hip/hip_runtime.h"
#include "lc_kernel.hu"
#include "io.hpp"
#ifdef __CUDA_ARCH__
#define __const__  __constant__ const
#else
#define __const__ const
#endif


/// DATA /// 
__const__ double probs[7] = {0.01047, 0.03125, 0.12500, 0.50000, 0.25000, 0.06250, 0.020833};
__const__ uint8_t byterev[256] = { 
0x00, 0x80, 0x40, 0xc0, 0x20, 0xa0, 0x60, 0xe0, 0x10, 0x90, 0x50, 0xd0, 0x30, 0xb0, 0x70, 0xf0, 
0x08, 0x88, 0x48, 0xc8, 0x28, 0xa8, 0x68, 0xe8, 0x18, 0x98, 0x58, 0xd8, 0x38, 0xb8, 0x78, 0xf8, 
0x04, 0x84, 0x44, 0xc4, 0x24, 0xa4, 0x64, 0xe4, 0x14, 0x94, 0x54, 0xd4, 0x34, 0xb4, 0x74, 0xf4, 
0x0c, 0x8c, 0x4c, 0xcc, 0x2c, 0xac, 0x6c, 0xec, 0x1c, 0x9c, 0x5c, 0xdc, 0x3c, 0xbc, 0x7c, 0xfc, 
0x02, 0x82, 0x42, 0xc2, 0x22, 0xa2, 0x62, 0xe2, 0x12, 0x92, 0x52, 0xd2, 0x32, 0xb2, 0x72, 0xf2, 
0x0a, 0x8a, 0x4a, 0xca, 0x2a, 0xaa, 0x6a, 0xea, 0x1a, 0x9a, 0x5a, 0xda, 0x3a, 0xba, 0x7a, 0xfa, 
0x06, 0x86, 0x46, 0xc6, 0x26, 0xa6, 0x66, 0xe6, 0x16, 0x96, 0x56, 0xd6, 0x36, 0xb6, 0x76, 0xf6, 
0x0e, 0x8e, 0x4e, 0xce, 0x2e, 0xae, 0x6e, 0xee, 0x1e, 0x9e, 0x5e, 0xde, 0x3e, 0xbe, 0x7e, 0xfe, 
0x01, 0x81, 0x41, 0xc1, 0x21, 0xa1, 0x61, 0xe1, 0x11, 0x91, 0x51, 0xd1, 0x31, 0xb1, 0x71, 0xf1, 
0x09, 0x89, 0x49, 0xc9, 0x29, 0xa9, 0x69, 0xe9, 0x19, 0x99, 0x59, 0xd9, 0x39, 0xb9, 0x79, 0xf9, 
0x05, 0x85, 0x45, 0xc5, 0x25, 0xa5, 0x65, 0xe5, 0x15, 0x95, 0x55, 0xd5, 0x35, 0xb5, 0x75, 0xf5, 
0x0d, 0x8d, 0x4d, 0xcd, 0x2d, 0xad, 0x6d, 0xed, 0x1d, 0x9d, 0x5d, 0xdd, 0x3d, 0xbd, 0x7d, 0xfd, 
0x03, 0x83, 0x43, 0xc3, 0x23, 0xa3, 0x63, 0xe3, 0x13, 0x93, 0x53, 0xd3, 0x33, 0xb3, 0x73, 0xf3, 
0x0b, 0x8b, 0x4b, 0xcb, 0x2b, 0xab, 0x6b, 0xeb, 0x1b, 0x9b, 0x5b, 0xdb, 0x3b, 0xbb, 0x7b, 0xfb, 
0x07, 0x87, 0x47, 0xc7, 0x27, 0xa7, 0x67, 0xe7, 0x17, 0x97, 0x57, 0xd7, 0x37, 0xb7, 0x77, 0xf7, 
0x0f, 0x8f, 0x4f, 0xcf, 0x2f, 0xaf, 0x6f, 0xef, 0x1f, 0x9f, 0x5f, 0xdf, 0x3f, 0xbf, 0x7f, 0xff, 
};


/// HOST-DEVICE FUNCTIONS
__host__ __device__ uint64_t extract_reverse_bits(uint8_t* data, uint64_t start, uint64_t bits){
    uint64_t end = start + bits; // [CHECK] -1?

    uint64_t s_byte = start / 8;
    uint64_t s_bit = start % 8;

    uint64_t e_byte = end / 8;
    uint64_t e_bit = end % 8;

    uint8_t right_bit_mask = (1 << (e_bit)) - 1;

    uint64_t output = 0;
    uint64_t i;
    for (i = e_byte; i >= s_byte; i--){
        uint8_t current_byte = byterev[data[i]];
        current_byte &= (i == e_byte) ? right_bit_mask : 0xff;
        current_byte >>= (i == s_byte) ? (s_bit) : 0;
        output <<= (i == s_byte) ? (8-s_bit) : 8;
        output |= current_byte;
        if(i == s_byte) break;
    }
    return output;
}

__host__ __device__ uint64_t parity(uint64_t num) {
    #ifdef  __CUDA_ARCH__
        return (__popcll(num) & 1);
    #else
        return __builtin_parityll(num);
    #endif
}

__host__ __device__ uint64_t trailing_zeros(uint64_t num) {
    #ifdef  __CUDA_ARCH__
        return __ffsll(num) - 1;
    #else
        return __builtin_ctzll(num);
    #endif
}

__host__ __device__ uint64_t complexity(uint64_t sequence, uint64_t bits) {
    if(sequence == 0){
        return 1;
    }
    uint64_t N = bits;
    uint64_t k = trailing_zeros(sequence);
    uint64_t F = (1 << (k + 1)) | 1;
    uint64_t G = 1;
    uint64_t l = k + 1;
    uint64_t a = k;
    uint64_t b = 0;
    uint64_t n = k;

    for(n = k + 1; n < N; n++){
        uint64_t d = parity(sequence & (F << (n - l)));
        if(d == 0){
            b+=1;
        }
        else{
            if(2*l > n){
                F ^= (G << (a - b));
                b += 1;
            }
            else{
                uint64_t T = F;
                F = (F << (b - a)) ^ G;
                l = n + 1 - l;
                G = T;
                a = b;
                b = n - l + 1;
            }
        }
    }

    return l;
}

__host__ __device__ double lc_test(uint8_t* data, uint64_t data_size, uint64_t bits) {
    uint64_t bins[7] = {0, 0, 0, 0, 0, 0, 0};
    uint64_t sequences_num = (data_size * 8) / bits;
    uint64_t i;

    double s_one = (bits & 1) ? -1.0 : 1.0;
    double mi = (double) (bits / 2.0);
    mi += (9.0 - s_one) / 36.0;
    mi -= ((bits / 3.0) + (2.0 / 9.0)) / pow(2.0, bits);

    for (i = 0; i < sequences_num; i++) {
        uint64_t starting_bit = i * bits;
        
        uint64_t sequence = extract_reverse_bits(data, starting_bit, bits);
        uint64_t lc = complexity(sequence, bits);

        double ti = s_one * ((double) lc - mi) + 2.0 / 9.0;
        bins[0] += (ti <= -2.5) ? 1u : 0;
        bins[1] += (ti > -2.5 && ti <= -1.5) ? 1u : 0;
        bins[2] += (ti > -1.5 && ti <= -0.5) ? 1u : 0;
        bins[3] += (ti > -0.5 && ti <= 0.5) ? 1u : 0;
        bins[4] += (ti > 0.5 && ti <= 1.5) ? 1u : 0;
        bins[5] += (ti > 1.5 && ti <= 2.5) ? 1u : 0;
        bins[6] += (ti > 2.5) ? 1u : 0;
    }

    double chi = 0.0;
    for (i = 0; i < 7; i++) {
        double expected = probs[i] * sequences_num;
        double enumerator = pow((double) bins[i] - expected, 2.0);
        chi += enumerator / expected;
    }
    
    return chi;
    
}


/// KERNEL
__global__ void lc_kernel(
    const uint8_t *data,
    uint64_t data_num, 
    uint64_t data_size,
    uint64_t bits,
    double*__restrict data_out,
    uint64_t thread_offset) {

    uint64_t thread_id = thread_offset + (blockIdx.x * blockDim.x) + threadIdx.x;
    if (thread_id < data_num){
        uint64_t byte_offset = thread_id * data_size;
        uint8_t* thred_data = (uint8_t*)&data[byte_offset];
        data_out[thread_id] = lc_test(thred_data, data_size, bits);
    }
}


/// GPU LAUNCHER
std::vector<double> run_gpu_lc_tests(
    const std::vector<uint8_t> &data,
    uint64_t data_num,
    uint64_t data_size,
    uint64_t bits) {

    uint64_t threads_per_block = 256;
    uint64_t blocks_per_grid = (data_num + threads_per_block - 1) / threads_per_block;

    uint8_t* dev_data_in;
    double* dev_data_out;
    hipMalloc((void**)&dev_data_in, data_num * data_size * sizeof(uint8_t));
    hipMalloc((void**)&dev_data_out, data_num * sizeof(double));

    hipMemcpy(dev_data_in, data.data(), data_num * data_size * sizeof(uint8_t), hipMemcpyHostToDevice);

    lc_kernel<<<blocks_per_grid, threads_per_block>>>(dev_data_in, data_num, data_size, bits, dev_data_out);

    std::vector<double> data_out(data_num);
    hipMemcpy(data_out.data(), dev_data_out, data_num * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_data_in);
    hipFree(dev_data_out);

    return data_out;
}


/// CPU LAUNCHER
std::vector<double> run_cpu_lc_tests(
    const std::vector<uint8_t> &data,
    uint64_t data_num,
    uint64_t data_size,
    uint64_t bits) {
    std::vector<double> data_out(data_num);
    for (uint64_t i = 0; i < data_num; i++) {
        uint64_t byte_offset = i * data_size;
        uint8_t* thred_data = (uint8_t*)&data[byte_offset];
        data_out[i] = lc_test(thred_data, data_size, bits);
    }
    return data_out;
}


/// PERFORMANCE TESTS
int lc_perf(uint64_t data_num, uint64_t data_size, uint64_t bits) {

    // DATA GENERATION
    hipEvent_t gen_start, gen_stop;
    float gen_time;
    hipEventCreate(&gen_start);
    hipEventRecord(gen_start, 0);

    std::vector<uint8_t> data_pieces(data_num * data_size);
    uint8_t r = 117;
    for (uint64_t i = 0; i < data_num * data_size; i++) {
        r = (r * r) + 117 * r + 17 + i;
        data_pieces[i] = r;
    }

    hipEventCreate(&gen_stop);
    hipEventRecord(gen_stop, 0);
    hipEventSynchronize(gen_stop);
    hipEventElapsedTime(&gen_time, gen_start, gen_stop);
    std::cout << "Data generation time: " << gen_time << " ms" << std::endl;

    // GPU TEST
    hipEvent_t gpu_start, gpu_stop;
    float gpu_time;
    hipEventCreate(&gpu_start);
    hipEventRecord(gpu_start, 0);

    std::vector<double> gpu_out = run_gpu_lc_tests(data_pieces, data_num, data_size, bits);

    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop);
    std::cout << "GPU time: " << gpu_time << " ms" << std::endl;

    // CPU TEST
    hipEvent_t cpu_start, cpu_stop;
    float cpu_time;
    hipEventCreate(&cpu_start);
    hipEventRecord(cpu_start, 0);

    std::vector<double> cpu_out = run_cpu_lc_tests(data_pieces, data_num, data_size, bits);

    hipEventCreate(&cpu_stop);
    hipEventRecord(cpu_stop, 0);
    hipEventSynchronize(cpu_stop);
    hipEventElapsedTime(&cpu_time, cpu_start, cpu_stop);
    std::cout << "CPU time: " << cpu_time << " ms" << std::endl;

    // CHECK
    for (uint64_t i = 0; i < data_num; i++) {
        if (abs(gpu_out[i] - cpu_out[i]) > 0.0001){
            std::cout << "ERROR: " << i << " " << gpu_out[i] << " " << cpu_out[i] << std::endl;
        }
    }

    std::cout << "GPU speedup: " << cpu_time / gpu_time << std::endl;
    return 0;
}
