#include "hip/hip_runtime.h"
#include "linear_complexity.hpp"
#include "io.hpp"
#include "lc_kernel.hu"
#include <iostream>
#include <vector>
#include <assert.h>


int main() {
  int files = 1024 * 1024; // 64k files
  int file_size = 1024; // 16KB = 128Kbits


  // DATA GENERATION
  hipEvent_t gen_start, gen_stop;
  float gen_time;

  hipEventCreate(&gen_start);
  hipEventRecord(gen_start, 0);

  std::vector<std::vector<uint8_t>> data_pieces(files);
  for(auto &piece : data_pieces) {
    piece.resize(file_size);
    for(int i = 0; i < file_size; i++) {
      piece[i] = rand() % 256;
    }
  }

  std::vector<uint8_t> data;
  for(auto &piece : data_pieces) {
    data.insert(data.end(), piece.begin(), piece.end());
  }

  hipEventCreate(&gen_stop);
  hipEventRecord(gen_stop, 0);
  hipEventSynchronize(gen_stop);
  hipEventElapsedTime(&gen_time, gen_start, gen_stop);
  std::cout << "Data generation time: " << gen_time << " ms" << std::endl;
  

  std::vector<double> dev_results(files);
  std::vector<double> host_results(files);

  // GPU
  hipEvent_t dev_start, dev_stop;
  float dev_elapsedTime;

  hipEventCreate(&dev_start);
  hipEventRecord(dev_start,0);

  run_lc_tests(data.data(), files, file_size, 31, dev_results.data());

  hipEventCreate(&dev_stop);
  hipEventRecord(dev_stop,0);
  hipEventSynchronize(dev_stop);

  hipEventElapsedTime(&dev_elapsedTime, dev_start,dev_stop);
  printf("Device time: %f ms\n", dev_elapsedTime);

  // CPU
  hipEvent_t host_start, host_stop;
  float host_elapsedTime;

  hipEventCreate(&host_start);
  hipEventRecord(host_start,0);

  for(int i = 0; i < files; i++) {
    host_results[i] = lc_test(data_pieces[i], 31);
  }

  hipEventCreate(&host_stop);
  hipEventRecord(host_stop,0);
  hipEventSynchronize(host_stop);

  hipEventElapsedTime(&host_elapsedTime, host_start,host_stop);
  printf("Host time: %f ms\n", host_elapsedTime);

  // Compare
  for(int i = 0; i < files; i++) {
    assert(abs(dev_results[i] - host_results[i]) < 0.0001);
  }

  printf("Success!\n");

  return 0;
}